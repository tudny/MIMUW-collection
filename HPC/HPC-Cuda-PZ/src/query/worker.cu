#include "hip/hip_runtime.h"
#include "worker.h"
#include "../logger.h"

#define BLOCK_SIZE 1024

int64_t highest_bit(int64_t x) {
    int64_t log = 0;
    for (; x; x >>= 1, log++);
    return log;
}

__global__ void query_index_on_gpu_linear_kernel(
        const IndexHash *indexes,
        const IndexHash *queries,
        QueryResult *results,
        size_t indexes_size,
        size_t queries_size
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= queries_size) {
        return;
    }

    IndexHash query = queries[idx];
    QueryResult result{NO_RESULT};

    for (size_t i = 0; i < indexes_size; i++) {
        IndexHash index = indexes[i];
        if (index == query) {
            result.offset_idx = i;
            break;
        }
    }

    results[idx] = result;
}

void query_index_on_gpu_linear(
        const std::shared_ptr<std::vector<IndexHash>> &indexes,
        const QueryList &queries,
        std::vector<QueryResult> &results
) {
    void *device_indexes;
    void *device_queries;
    void *device_results;
    size_t indexes_size = indexes->size() * sizeof(IndexHash);
    size_t queries_size = queries.size() * sizeof(IndexHash);
    size_t results_size = queries.size() * sizeof(QueryResult);

    hipMalloc(&device_indexes, indexes_size);
    hipMalloc(&device_queries, queries_size);
    hipMalloc(&device_results, results_size);

    hipMemcpy(device_indexes, indexes->data(), indexes_size, hipMemcpyHostToDevice);
    hipMemcpy(device_queries, queries.data(), queries_size, hipMemcpyHostToDevice);

    size_t blocks = (queries.size() + BLOCK_SIZE - 1) / BLOCK_SIZE;
    query_index_on_gpu_linear_kernel<<<blocks, BLOCK_SIZE>>>(
            (IndexHash *) device_indexes,
            (IndexHash *) device_queries,
            (QueryResult *) device_results,
            indexes->size(),
            queries.size()
    );

    hipMemcpy(results.data(), device_results, results_size, hipMemcpyDeviceToHost);

    hipFree(device_indexes);
    hipFree(device_queries);
    hipFree(device_results);
}

__global__ void query_index_on_gpu_binary_kernel(
        const IndexHash *indexes,
        const IndexHash *queries,
        QueryResult *results,
        size_t indexes_size,
        size_t queries_size
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= queries_size) {
        return;
    }

    IndexHash query = queries[idx];
    QueryResult result = {NO_RESULT};

    int64_t left = -1;
    int64_t right = (int) indexes_size;
    while (left + 1 < right) {
        int64_t mid = (left + right) / 2;
        IndexHash index = indexes[mid];
        if (index < query) {
            left = mid;
        } else if (index > query) {
            right = mid;
        } else {
            result.offset_idx = mid;
            break;
        }
    }

    results[idx] = result;
}

void query_index_on_gpu_binary(
        const std::shared_ptr<std::vector<IndexHash>> &indexes,
        const QueryList &queries,
        std::vector<QueryResult> &results
) {
    void *device_indexes;
    void *device_queries;
    void *device_results;
    size_t indexes_size = indexes->size() * sizeof(IndexHash);
    size_t queries_size = queries.size() * sizeof(IndexHash);
    size_t results_size = queries.size() * sizeof(QueryResult);

    hipMalloc(&device_indexes, indexes_size);
    hipMalloc(&device_queries, queries_size);
    hipMalloc(&device_results, results_size);

    TIMEIT("Index host->GPU", hipMemcpy(device_indexes, indexes->data(), indexes_size, hipMemcpyHostToDevice));
    TIMEIT("Query host->GPU", hipMemcpy(device_queries, queries.data(), queries_size, hipMemcpyHostToDevice));

    uint64_t blocks = ((uint64_t) queries.size() + BLOCK_SIZE - 1) / BLOCK_SIZE;
    TIMEIT_BEGIN(kernel);
    query_index_on_gpu_binary_kernel<<<blocks, BLOCK_SIZE>>>(
            (IndexHash *) device_indexes,
            (IndexHash *) device_queries,
            (QueryResult *) device_results,
            indexes->size(),
            queries.size()
    );
    TIMEIT_END(kernel);

    TIMEIT("Result GPU->host", hipMemcpy(results.data(), device_results, results_size, hipMemcpyDeviceToHost));

    hipFree(device_indexes);
    hipFree(device_queries);
    hipFree(device_results);
}

__global__ void query_index_on_gpu_binary_kernel_oneway(
        const IndexHash *indexes,
        const IndexHash *queries,
        QueryResult *results,
        size_t indexes_size,
        size_t queries_size,
        int64_t log
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= queries_size) {
        return;
    }
    IndexHash query = queries[idx];

    int64_t begin = -1;
    for (int64_t step = 1 << log; step >= 1; step >>= 1) {
        int64_t next = begin + step;
        if (next < (int64_t) indexes_size && (query >= indexes[next])) {
            begin = next;
        }
    }

    results[idx].offset_idx = NO_RESULT;
    if (begin != -1) {
        IndexHash index = indexes[begin];
        if (index == query) {
            results[idx].offset_idx = begin;
        }
    }
}

void query_index_on_gpu_binary_oneway(
        const std::shared_ptr<std::vector<IndexHash>> &indexes,
        const QueryList &queries,
        std::vector<QueryResult> &results
) {
    void *device_indexes;
    void *device_queries;
    void *device_results;
    size_t indexes_size = indexes->size() * sizeof(IndexHash);
    size_t queries_size = queries.size() * sizeof(IndexHash);
    size_t results_size = queries.size() * sizeof(QueryResult);

    hipMalloc(&device_indexes, indexes_size);
    hipMalloc(&device_queries, queries_size);
    hipMalloc(&device_results, results_size);

    TIMEIT("Index host->GPU", hipMemcpy(device_indexes, indexes->data(), indexes_size, hipMemcpyHostToDevice));
    TIMEIT("Query host->GPU", hipMemcpy(device_queries, queries.data(), queries_size, hipMemcpyHostToDevice));

    int64_t blocks = ((int64_t) queries.size() + BLOCK_SIZE - 1) / BLOCK_SIZE;
    TIMEIT_BEGIN(kernel);
    query_index_on_gpu_binary_kernel_oneway<<<blocks, BLOCK_SIZE>>>(
            (IndexHash *) device_indexes,
            (IndexHash *) device_queries,
            (QueryResult *) device_results,
            indexes->size(),
            queries.size(),
            highest_bit((int) indexes->size())
    );
    TIMEIT_END(kernel);

    TIMEIT("Result GPU->host", hipMemcpy(results.data(), device_results, results_size, hipMemcpyDeviceToHost));

    hipFree(device_indexes);
    hipFree(device_queries);
    hipFree(device_results);
}
