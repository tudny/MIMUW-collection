#include "hip/hip_runtime.h"
#include <cstdio>
#include "errors.h"

#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void transpose(float *odata, const float *idata)
{
    __shared__ float temp[(TILE_DIM + 1) * TILE_DIM];

    // (x, y) are coordinates inside the global matrix
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        temp[(threadIdx.y + j) * (TILE_DIM + 1) + threadIdx.x] = idata[(y + j) * width + x];
    }

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        odata[(y + j) * width + x] = temp[threadIdx.x * (TILE_DIM + 1) + threadIdx.y + j];
    }
}

// __global__ void transpose(float *odata, const float *idata)
// {
//   // (x, y) are coordinates inside the global matrix
//   int x = blockIdx.x * TILE_DIM + threadIdx.x;
//   int y = blockIdx.y * TILE_DIM + threadIdx.y;
//   int width = gridDim.x * TILE_DIM;

//   for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
//     odata[x*width + (y+j)] = idata[(y+j)*width + x];
//   }
// }

int main()
{
    const int nx = 8192;
    const int ny = 8192;

    const int mem_size = nx * ny * sizeof(float);

    dim3 dimGrid(nx / TILE_DIM, ny / TILE_DIM, 1);

    // Each block is responsible for transposing one TILE_DIM x TILE_DIM submatrix
    dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

    if (nx % TILE_DIM || ny % TILE_DIM)
    {
        printf("nx and ny must be a multiple of TILE_DIM\n");
        return 1;
    }

    if (TILE_DIM % BLOCK_ROWS)
    {
        printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
        return 1;
    }

    float *host_input = (float *)malloc(mem_size), *host_correct = (float *)malloc(mem_size), *host_output = (float *)malloc(mem_size);
    float *dev_input, *dev_output;

    HANDLE_ERROR(hipMalloc(&dev_input, mem_size));
    HANDLE_ERROR(hipMalloc(&dev_output, mem_size));

    for (int j = 0; j < ny; j++)
        for (int i = 0; i < nx; i++)
            host_input[j * nx + i] = host_correct[i * nx + j] = j * nx + i;

    HANDLE_ERROR(hipMemcpy(dev_input, host_input, mem_size, hipMemcpyHostToDevice));

    hipEvent_t startEvent, stopEvent;
    HANDLE_ERROR(hipEventCreate(&startEvent));
    HANDLE_ERROR(hipEventCreate(&stopEvent));
    HANDLE_ERROR(hipEventRecord(startEvent, 0));

    transpose<<<dimGrid, dimBlock>>>(dev_output, dev_input);

    HANDLE_ERROR(hipEventRecord(stopEvent, 0));
    HANDLE_ERROR(hipEventSynchronize(stopEvent));

    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));
    printf("Time to generate: %3.1f ms\n", elapsedTime);

    HANDLE_ERROR(hipMemcpy(host_output, dev_output, mem_size, hipMemcpyDeviceToHost));

    for (int j = 0; j < ny; j++)
        for (int i = 0; i < nx; i++)
            if (host_output[j * nx + i] != host_correct[j * nx + i])
            {
                printf("Wrong value at (%d, %d), got %f, expected %f\n", i, j, host_output[j * nx + i], host_correct[j * nx + i]);
                return 1;
            }
}
