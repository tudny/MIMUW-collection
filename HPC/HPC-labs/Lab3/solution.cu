
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#ifndef RADIUS
#define RADIUS -1
#endif

#ifndef NUM_ELEMENTS
#define NUM_ELEMENTS -1
#endif

#define MAX_NUM 2500

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))

static void handleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define cudaCheck(err) (handleError(err, __FILE__, __LINE__))

__global__ void stencil_1d(int *in, int *out)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i > NUM_ELEMENTS)
        return;
    int start = MAX(i - RADIUS, 0);
    int end = MIN(i + RADIUS, NUM_ELEMENTS - 1);
    int result = 0;
    for (int k = start; k <= end; ++k)
    {
        result += in[k];
    }
    out[i] = result;
}

void cpu_slow_stencil_1d(int *in, int *out)
{
    #pragma omp parallel for
    for (int i = 0; i < NUM_ELEMENTS; ++i)
    {
        out[i] = 0;
        int start = MAX(i - RADIUS, 0);
        int end = MIN(i + RADIUS, NUM_ELEMENTS - 1);
        for (int k = start; k <= end; ++k)
        {
            out[i] += in[k];
        }
    }
}

void cpu_stencil_1d(int *in, int *out)
{
    int current_sum = 0;
    for (size_t i = 0; i < RADIUS; ++i)
    {
        current_sum += in[i];
    }

    int idxToRemove = -1 - RADIUS - 1;
    int idxToAdd = -1 + RADIUS;
    for (size_t i = 0; i < NUM_ELEMENTS; ++i)
    {
        ++idxToRemove;
        ++idxToAdd;
        if (idxToRemove >= 0)
        {
            current_sum -= in[idxToRemove];
        }
        if (idxToAdd < NUM_ELEMENTS)
        {
            current_sum += in[idxToAdd];
        }
        out[i] = current_sum;
    }
}

int main()
{
    // ************
    // MEMORY SETUP
    // ************

    int *in = (int *)malloc(NUM_ELEMENTS * sizeof(int));
    int *out = (int *)malloc(NUM_ELEMENTS * sizeof(int));
    int *out_cuda_but_on_host = (int *)malloc(NUM_ELEMENTS * sizeof(int));

    for (size_t i = 0; i < NUM_ELEMENTS; ++i)
    {
        in[i] = rand() % MAX_NUM;
    }

    int *cuda_in;
    int *cuda_out;

    // ************
    // SETUP CUDA MEMORY
    // ************

    hipEvent_t start, stop, start_copy, end_copy, start_kernel, end_kernel, start_restore, end_restore;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start_copy);
    hipEventCreate(&end_copy);
    hipEventCreate(&start_kernel);
    hipEventCreate(&end_kernel);
    hipEventCreate(&start_restore);
    hipEventCreate(&end_restore);

    hipEventRecord(start, 0);

    hipEventRecord(start_copy, 0);
    cudaCheck(hipMalloc((void **)&cuda_in, NUM_ELEMENTS * sizeof(int)));
    cudaCheck(hipMalloc((void **)&cuda_out, NUM_ELEMENTS * sizeof(int)));

    // ************
    // COPY TO GPU
    // ************

    cudaCheck(hipMemcpy(cuda_in,
                         in,
                         NUM_ELEMENTS * sizeof(int),
                         hipMemcpyHostToDevice));

    hipEventRecord(end_copy, 0);

    // ************
    // EXECUTE KERNEL
    // ************

    int blockSize = 256;
    int gridSize = (NUM_ELEMENTS + blockSize - 1) / blockSize;

    hipEventRecord(start_kernel, 0);
    stencil_1d<<<gridSize, blockSize>>>(cuda_in, cuda_out);
    hipEventRecord(end_kernel, 0);

    cudaCheck(hipPeekAtLastError());

    // ************
    // COPY RESULT BACK
    // ************

    hipEventRecord(start_restore, 0);
    cudaCheck(hipMemcpy(out_cuda_but_on_host,
                         cuda_out,
                         NUM_ELEMENTS * sizeof(int),
                         hipMemcpyDeviceToHost));
    hipEventRecord(end_restore, 0);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    float elapsedCopy;
    float elapsedKernel;
    float elapsedRestore;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventElapsedTime(&elapsedCopy, start_copy, end_copy);
    hipEventElapsedTime(&elapsedKernel, start_kernel, end_kernel);
    hipEventElapsedTime(&elapsedRestore, start_restore, end_restore);
    printf("Total GPU execution time:  %3.1f ms\n", elapsedTime);
    printf("Total GPU init copy time:  %3.1f ms\n", elapsedCopy);
    printf("Total GPU kernel exe time: %3.1f ms\n", elapsedKernel);
    printf("Total GPU restore time:    %3.1f ms\n", elapsedRestore);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start_copy);
    hipEventDestroy(end_copy);
    hipEventDestroy(start_kernel);
    hipEventDestroy(end_kernel);
    hipEventDestroy(start_restore);
    hipEventDestroy(end_restore);

    // ************
    // FREE CUDA MEMORY
    // ************

    cudaCheck(hipFree(cuda_in));
    cudaCheck(hipFree(cuda_out));

    // ************
    // RUN CPU
    // ************

    struct timespec cpu_start, cpu_stop;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);

    cpu_slow_stencil_1d(in, out);

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
    double result = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
    printf("CPU execution time:  %3.1f ms\n", result);


    // ************
    // RUN CPU FAST
    // ************

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);

    cpu_stencil_1d(in, out);

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
    result = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
    printf("CPU FAST execution time:  %3.1f ms\n", result);


    // ************
    // SANITY CHECK
    // ************

    for (size_t i = 0; i < NUM_ELEMENTS; ++i)
    {
        if (out[i] != out_cuda_but_on_host[i])
        {
            printf("FAIL: %ld\n", i);
            printf("Expected %d, but got %d\n", out[i], out_cuda_but_on_host[i]);
            break;
        }
    }

    free(in);
    free(out);
    free(out_cuda_but_on_host);

    return 0;
}
